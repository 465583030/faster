#include <iostream>
#include "libfaster.h"

#define NUMITEMS 10*1000

using namespace std;
using namespace faster;

typedef int (* map1_t)(int &);

__device__ int map1(int & input){
	return input / 2;
}

int reduce1(int &a, int &b){
	return a + b;
}



int main(int argc, char ** argv){
	// Init Faster Framework
	cout << "Init FastLib" << '\n';
	fastContext fc(argc,argv);

	__device__ map1_t map1_d = & map1;
	map1_t map1_h;
	hipMemcpyFromSymbol(&map1_h, HIP_SYMBOL(map1_d), sizeof(pointFunction_t));
	fc.registerFunction((void*) &map1_h);
	fc.registerFunction((void*) &reduce1);

	fc.startWorkers();
	if (!fc.isDriver())
		return 0;

	cout << "Generate Data" << '\n';
	int rawdata[NUMITEMS];

	for ( int i = 0; i < NUMITEMS; ++i )
		rawdata[i] = 2;

	cout << "Import Data" << '\n';
	fdd <int> data(fc, rawdata, NUMITEMS);

	cout << "Process Data" << '\n';
	int result = data.map<int>(&map1_h)->reduce(&reduce1);

	cout << "DONE!" << '\n';

	std::cout << "Resut:" << result << "\n";

	return 0;
}
